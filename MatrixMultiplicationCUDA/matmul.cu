#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 16

void matMulCPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    int i, j, k;
    int offsetA, offsetB;
    float cumSum;

    for (i = 0; i < numARows; i++) {
        for (j = 0; j < numBCols; j++) {
            cumSum = 0;
            for (k = 0; k < numACols; k++) {
                // linearize index
                offsetA = i*numACols + k;
                offsetB = k*numBCols + j;

                // accumulate element-wise product
                cumSum += A[offsetA] * B[offsetB];
            }
            C[i*numBCols + j] = cumSum;
        }
    }
}

__global__ void matMulGPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    // compute global thread coordinates
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // linearize coordinates for data access
    int offset = row * numBCols + col;

    if ((row < numARows) && (col < numBCols)) {
        float cumSum = 0;
        for (int k = 0; k < numACols; k++) {
            cumSum += A[row*numACols + k] * B[k*numBCols + col];
        }
        C[offset] = cumSum;
    }
}


int main(int argc, char* argv[]) {
    int numARows = 960;
    int numACols = 640;
    int numBCols = 800;

    if (argc ==4 ) {
        numARows = std::stoi(argv[1]);
        numACols = std::stoi(argv[2]);
        numBCols = std::stoi(argv[3]);
    }

    cout<<"Size of matrix A: "<<numARows<<" "<<numACols<<endl;
    cout<<"Size of matrix B: "<<numACols<<" "<<numBCols<<endl;    

    // timers
    GpuTimer timer0, timer1, timer2, timer3;

    size_t sizeA = numARows * numACols * sizeof(float);
    size_t sizeB = numACols * numBCols * sizeof(float);
    size_t sizeC = numARows * numBCols * sizeof(float);


    // allocate host memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);
    float* h_C_CPU = (float*)malloc(sizeC);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  numARows; i++) {
        for (j = 0; j < numACols; j++) {
            offset = i*numACols + j;
            h_A[offset] = sin(i);
        }
    }
    for (i = 0; i <  numACols; i++) {
        for (j = 0; j < numBCols; j++) {
            offset = i*numBCols + j;
            h_B[offset] = cos(j);
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    timer0.Start();
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);
    timer0.Stop();
    printf("Time to allocate memory on the device is: %f msecs.\n", timer0.Elapsed());

    // transfer to GPU
    timer1.Start();
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);
    timer1.Stop();
    printf("Time to copy the Matrix from host to device is: %f msecs.\n", timer1.Elapsed());

    // kernel launch
    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blockPerGrid(ceil(numBCols/(float)BLOCK_SIZE), ceil(numACols/(float)BLOCK_SIZE), 1);
    timer2.Start();
    matMulGPU<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C, numARows, numACols, numBCols);
    timer2.Stop();
    printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

    // transfer to CPU
    timer3.Start();
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);
    timer3.Stop();
    printf("Time to copy the resulting Matrix from device to host is: %f msecs.\n", timer3.Elapsed());

    clock_t begin = clock();
    matMulCPU(h_A, h_B, h_C_CPU, numARows, numACols, numBCols);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
    printf("Implemented CPU code ran in: %f msecs.\n", time_spent);

    free(h_A); free(h_B); free(h_C); free(h_C_CPU);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    printf("Done");

    return 0;
}