#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 16
#define TILE_WIDTH 16

void matMulCPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    int i, j, k;
    int offsetA, offsetB;
    float cumSum;

    for (i = 0; i < numARows; i++) {
        for (j = 0; j < numBCols; j++) {
            cumSum = 0;
            for (k = 0; k < numACols; k++) {
                // linearize index
                offsetA = i*numACols + k;
                offsetB = k*numBCols + j;

                // accumulate element-wise product
                cumSum += A[offsetA] * B[offsetB];
            }
            C[i*numBCols + j] = cumSum;
        }
    }
}

__global__ void matMulGPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    // allocate shared memory
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    // coordinates for C
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float cumSum = 0;
    for (int m = 0; m < ceil(numACols/(float)TILE_WIDTH); m++) {
        // load tiles
        if ((row < numARows) && ((m*TILE_WIDTH + tx) < numACols))
            sharedA[ty][tx] = A[row*numACols + m*TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0;
        if ((col < numBCols) && ((m*TILE_WIDTH + ty) < numACols))
            sharedB[ty][tx] = B[(m*TILE_WIDTH + ty)*numBCols + col];
        else
            sharedB[ty][tx] = 0;
        // pause until all threads have loaded tile values
        __syncthreads();

        // compute partial dot product (for individual thread)
        for (int k = 0; k < TILE_WIDTH; k++) {
            cumSum += sharedA[ty][k] * sharedB[k][tx];
        }
        // wait until all threads have used tile values
        __syncthreads();
    }
    if((row < numACols) && (col < numBCols)) {
        C[row*numBCols + col] = cumSum;
    }
}


int main(int argc, char* argv[]) {
    int numARows = 960;
    int numACols = 640;
    int numBCols = 800;

    if (argc == 4) {
        numARows = std::stoi(argv[1]);
        numACols = std::stoi(argv[2]);
        numBCols = std::stoi(argv[3]);
    }

    cout<<"Size of matrix A: "<<numARows<<" "<<numACols<<endl;
    cout<<"Size of matrix B: "<<numACols<<" "<<numBCols<<endl;

    // timers
    GpuTimer timer0, timer1, timer2, timer3;

    size_t sizeA = numARows * numACols * sizeof(float);
    size_t sizeB = numACols * numBCols * sizeof(float);
    size_t sizeC = numARows * numBCols * sizeof(float);

    // allocate host memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);
    float* h_C_CPU = (float*)malloc(sizeC);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  numARows; i++) {
        for (j = 0; j < numACols; j++) {
            offset = i*numACols + j;
            h_A[offset] = sin(i);
        }
    }
    for (i = 0; i <  numACols; i++) {
        for (j = 0; j < numBCols; j++) {
            offset = i*numBCols + j;
            h_B[offset] = cos(j);
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    timer0.Start();
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);
    timer0.Stop();
    printf("Time to allocate memory on the device is: %f msecs.\n", timer0.Elapsed());

    // transfer to GPU
    timer1.Start();
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);
    timer1.Stop();
    printf("Time to copy the Matrix from host to device is: %f msecs.\n", timer1.Elapsed());

    // kernel launch
    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blockPerGrid(ceil(numBCols/(float)BLOCK_SIZE), ceil(numACols/(float)BLOCK_SIZE), 1);
    timer2.Start();
    matMulGPU<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C, numARows, numACols, numBCols);
    timer2.Stop();
    printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

    // transfer to CPU
    timer3.Start();
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);
    timer3.Stop();
    printf("Time to copy the resulting Matrix from device to host is: %f msecs.\n", timer3.Elapsed());

    clock_t begin = clock();
    matMulCPU(h_A, h_B, h_C_CPU, numARows, numACols, numBCols);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
    printf("Implemented CPU code ran in: %f msecs.\n", time_spent);

    free(h_A); free(h_B); free(h_C); free(h_C_CPU);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}